#include "hip/hip_runtime.h"
// collatz_gpu_mpi_cycles.cu
// MPI + CUDA Collatz scanner with end-of-run cycle printing on rank 0.
//
// Build:
//   nvcc -O3 -arch=sm_90 --device-int128 collatz_gpu_mpi_cycles.cu -o collatz_gpu_mpi_cycles -lmpi
//
// Notes:
// - One MPI rank per GPU is the simplest setup (srun -n <num_gpus_total>).
// - The kernel scans candidates (skipping evens & multiples of 3) and pushes flagged seeds
//   (128-bit overflow or fuse-hit) to a device queue.
// - After the kernel, the host verifies flags with Brent(128), and if needed Brent(256) on CPU.
// - Only rank 0 prints a final summary; with --print-cycles it also prints each cycle line.
//
// CLI:
//   ./collatz_gpu_mpi_cycles <start_offset> <count> [--small-limit <bits>] [--tag <name>] [--no-print-cycles]
//
//   <start_offset>, <count> : range within [2^71 + start_offset, 2^71 + start_offset + count)
//   --small-limit <bits>    : memo table size as 2^bits (default: 20 → ~4MB)
//   --tag <name>            : tag (currently only used in headers; logging to files removed for clarity)
//   --no-print-cycles       : do not print the individual cycle lines (summary still prints)
//
// -------------------------------------------------------------------------

#include <mpi.h>
#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdint>
#include <cstring>
#include <vector>
#include <string>
#include <algorithm>
#include <iostream>
#include <cassert>

// ---------------- Common typedefs & constants ----------------
using u128 = unsigned __int128;

static constexpr uint64_t SAFETY_FUSE   = 100000;   // fast path fuse
static constexpr uint64_t EXTENDED_FUSE = 1000000;  // verification fuse
static constexpr uint32_t UNKNOWN       = UINT32_MAX;

// ---------------- Helpers: u128 (host & device) ----------------
__host__ __device__ static inline u128 u128_from_u64(uint64_t v){ return (u128)v; }

// Replace your current ctz_u128 with this pair:

// 64-bit trailing-zero count that works in both host & device builds
__host__ __device__ static inline int ctz64_u(uint64_t x) {
#ifdef __CUDA_ARCH__
    // device: __ffsll returns 1 + index of LSB set; subtract 1
    return __ffsll((long long)x) - 1;
#else
    // host: built-in works here
    return __builtin_ctzll(x);
#endif
}

__host__ __device__ static inline int ctz_u128(u128 x) {
    if (x == 0) return 128;
    uint64_t lo = (uint64_t)x;
    if (lo) return ctz64_u(lo);
    uint64_t hi = (uint64_t)(x >> 64);
    return 64 + ctz64_u(hi);
}
__host__ __device__ static inline u128 max_safe_u128() {
    // ((~(u128)0) - 1) / 3
    return ( ( (u128)~(u128)0 ) - 1 ) / 3;
}
// Convert u128 to decimal string (host only)
// Note: for cycle printing only; not performance critical
static std::string to_string_u128(u128 v) {
    if (v == 0) return "0";
    std::string s; s.reserve(40);
    while (v) {
        int d = (int)(v % 10);
        s.push_back('0' + d);
        v /= 10;
    }
    std::reverse(s.begin(), s.end());
    return s;
}


// ---------------- Device memo table ----------------
struct DevMemo {
    const uint32_t* tbl;
    uint64_t limit;
};

__device__ __forceinline__ uint32_t memo_lookup(const DevMemo& M, u128 x){
    // If x fits in 64 bits and < limit, check memo
    uint64_t v = (uint64_t)x;
    if ((u128)v == x && v < M.limit){
        uint32_t m = M.tbl[v];
        return m;
    }
    return UNKNOWN;
}

// ---------------- Device: fast-path compute (128-bit) ----------------
__device__ __forceinline__
void compute_collatz_readonly_gpu(u128 n, const DevMemo& M,
                                  uint64_t max_steps,
                                  uint64_t& steps_out,
                                  bool& overflow_out,
                                  u128& peak_out)
{
    steps_out = 0;
    overflow_out = false;
    peak_out = n;
    if (n <= 1) return;

    const u128 MAX_SAFE = max_safe_u128();
    u128 cur = n;

    while (cur != 1) {
        if (steps_out >= max_steps) return;

        if (cur < M.limit) {
            uint32_t cached = memo_lookup(M, cur);
            if (cached != UNKNOWN) {
                steps_out += cached;
                return;
            }
        }
        if (cur > peak_out) peak_out = cur;

        if ( ((uint64_t)cur & 1ULL) == 0ULL ){
            int sh = ctz_u128(cur);
            cur >>= sh;
            steps_out += sh;
        } else {
            if (cur > MAX_SAFE) { overflow_out = true; return; }
            u128 t = 3*cur + 1;
            int k = ctz_u128(t);
            cur = t >> k;
            steps_out += 1 + k;
        }
    }
}

// ---------------- Device: flagged queue ----------------
struct FlaggedSeed {
    u128     n;
    uint64_t steps_before;
    uint8_t  reason;  // 1=overflow, 2=fuse, 3=both (unlikely)
};

__device__ unsigned int g_q_count = 0;

__device__ void push_flagged(FlaggedSeed* q, unsigned int q_cap,
                             u128 n, uint64_t steps, uint8_t reason)
{
    unsigned int pos = atomicAdd(&g_q_count, 1);
    if (pos < q_cap){
        q[pos].n = n;
        q[pos].steps_before = steps;
        q[pos].reason = reason;
    }
    // else: drop (rare); could set a global overflow flag if desired
}

// ---------------- Device: kernel ----------------
struct ThreadAcc {
    uint64_t tested;
    uint64_t steps_sum;
    uint64_t overflow_cnt;
    uint64_t fuse_cnt;
};

__global__ void collatz_kernel(u128 start, u128 end, uint64_t first_delta,
                               DevMemo M,
                               FlaggedSeed* __restrict flag_q, unsigned int q_cap,
                               ThreadAcc* __restrict acc_out)
{
    const uint64_t tid = blockIdx.x * (uint64_t)blockDim.x + threadIdx.x;
    const uint64_t stride = gridDim.x * (uint64_t)blockDim.x;

    // Walk the 2/4 delta sequence to position this thread at its first candidate
    u128 n = start;
    uint64_t d = first_delta;
    for (uint64_t i=0;i<tid;i++){
        n += d;
        d ^= 6ULL; // 2 ↔ 4
    }

    // Accumulators
    uint64_t tested=0, steps_sum=0, over_c=0, fuse_c=0;

    // Grid-stride loop: for each "round", advance 'stride' positions in the 2/4 toggle
    for (;;) {
        if (n >= end) break;

        // compute
        uint64_t steps=0;
        bool overflow=false;
        u128 peak;
        compute_collatz_readonly_gpu(n, M, SAFETY_FUSE, steps, overflow, peak);

        tested++;
        steps_sum += steps;

        uint8_t reason = 0;
        if (overflow){ over_c++; reason |= 1; }
        if (steps >= SAFETY_FUSE){ fuse_c++; reason |= 2; }
        if (reason){
            push_flagged(flag_q, q_cap, n, steps, reason);
        }

        // advance by 'stride' toggles
        for (uint64_t s=0; s<stride; s++){
            n += d;
            d ^= 6ULL;
        }
    }

    // Write per-thread accumulators
    acc_out[tid].tested = tested;
    acc_out[tid].steps_sum = steps_sum;
    acc_out[tid].overflow_cnt = over_c;
    acc_out[tid].fuse_cnt = fuse_c;
}

// ---------------- Host: small memo table ----------------
// Minimal but correct base entries. You can graft a full precompute if desired.
static void build_min_memo(std::vector<uint32_t>& memo){
    if (memo.size() == 0) return;
    std::fill(memo.begin(), memo.end(), UNKNOWN);
    memo[0] = UNKNOWN;
    if (memo.size() > 1) memo[1] = 0;
    if (memo.size() > 2) memo[2] = 1;
    if (memo.size() > 3) memo[3] = 7;
}

// ---------------- Host: align start & delta (skip evens and multiples of 3) ----------------
static inline uint32_t mod3_u128(u128 n) {
    uint64_t lo = (uint64_t)n;
    uint64_t hi = (uint64_t)(n >> 64);
    uint32_t r_lo = lo % 3;
    uint32_t r_hi = hi % 3;
    return (r_lo + r_hi) % 3;
}

static inline void align_start_and_delta(u128 &n, uint64_t &delta) {
    if ( ((uint64_t)n & 1ULL) == 0ULL ) n += 1;
    uint32_t r3 = mod3_u128(n);
    if (r3 == 0) n += 2;
    r3 = mod3_u128(n);
    delta = (r3 == 1) ? 4 : 2;
}

// ---------------- Host: Brent(128) & Brent(256) ----------------
struct Brent128Res {
    bool cycle_found;
    uint64_t steps;
    uint64_t cycle_length;
    u128 meet_value;
    bool overflow;
};

static Brent128Res detect_cycle_brent_128_host(u128 n, uint64_t max_steps) {
    Brent128Res r{false,0,0,0,false};
    u128 tortoise = n, hare = n;
    uint64_t power = 1, lambda = 1, steps = 0;
    const u128 MAX_SAFE = max_safe_u128();

    while (steps < max_steps) {
        if (hare == 1) { r.steps = steps; return r; }

        if ( ((uint64_t)hare & 1ULL) == 0ULL ){
            int sh = ctz_u128(hare);
            hare >>= sh; steps += sh;
        } else {
            if (hare > MAX_SAFE){ r.overflow=true; r.steps=steps; return r; }
            u128 t = 3*hare + 1;
            int k = ctz_u128(t);
            hare = t >> k; steps += 1 + k;
        }
        if (tortoise == hare){
            r.cycle_found = true;
            r.cycle_length = lambda;
            r.steps = steps;
            r.meet_value = hare;
            return r;
        }
        if (lambda == power){
            tortoise = hare;
            power <<= 1;
            lambda = 0;
        }
        lambda++;
    }
    r.steps = steps; r.overflow = true;
    return r;
}

// Lightweight uint256 for host Brent(256)
struct u256 {
    uint64_t v[4]; // little-endian limbs: v[0] is lowest

    u256(){ v[0]=v[1]=v[2]=v[3]=0; }
    explicit u256(u128 x){
        v[0] = (uint64_t)x;
        v[1] = (uint64_t)(x >> 64);
        v[2] = v[3] = 0;
    }
    bool is_zero() const { return (v[0]|v[1]|v[2]|v[3])==0; }
    bool is_one()  const { return v[0]==1 && v[1]==0 && v[2]==0 && v[3]==0; }
    bool is_even() const { return (v[0] & 1ULL)==0; }
};

static inline int ctz_u256(const u256& a){
    if (a.v[0]) return __builtin_ctzll(a.v[0]);
    if (a.v[1]) return 64 + __builtin_ctzll(a.v[1]);
    if (a.v[2]) return 128 + __builtin_ctzll(a.v[2]);
    if (a.v[3]) return 192 + __builtin_ctzll(a.v[3]);
    return 256;
}
static inline bool eq_u256(const u256& a, const u256& b){
    return a.v[0]==b.v[0] && a.v[1]==b.v[1] && a.v[2]==b.v[2] && a.v[3]==b.v[3];
}
static inline bool gt_u256(const u256& a, const u256& b){
    for (int i=3;i>=0;i--){
        if (a.v[i] > b.v[i]) return true;
        if (a.v[i] < b.v[i]) return false;
    }
    return false;
}
static inline u256 shr_u256(const u256& a, int sh){
    if (sh<=0) return a;
    if (sh>=256) return u256();
    u256 r;
    int limb = sh/64, bits = sh%64;
    for (int i=0;i<4-limb;i++){
        uint64_t low = a.v[i+limb] >> bits;
        uint64_t hi  = 0;
        if (bits && i+limb+1<4) hi = a.v[i+limb+1] << (64-bits);
        r.v[i] = low | hi;
    }
    for (int i=4-limb;i<4;i++) r.v[i]=0;
    return r;
}
static inline u256 add_u256_u64(const u256& a, uint64_t b){
    u256 r=a;
    __uint128_t s = ( (__uint128_t)r.v[0] + b );
    r.v[0] = (uint64_t)s;
    uint64_t c = (uint64_t)(s>>64);
    for (int i=1;i<4 && c;i++){
        s = ( (__uint128_t)r.v[i] + c );
        r.v[i] = (uint64_t)s;
        c = (uint64_t)(s>>64);
    }
    return r;
}
static inline u256 mul_u256_u64(const u256& a, uint64_t m){
    u256 r;
    __uint128_t c=0;
    for (int i=0;i<4;i++){
        __uint128_t p = (__uint128_t)a.v[i]*m + c;
        r.v[i] = (uint64_t)p;
        c = p>>64;
    }
    return r;
}
static inline bool would_overflow_3n_plus_1_u256(const u256& a){
    // Keep same MAX_SAFE_256 as user code (~0x5555... pattern)
    static const u256 LIMIT = []{
        u256 t; t.v[0]=0x5555555555555555ULL;
                t.v[1]=0x5555555555555555ULL;
                t.v[2]=0x5555555555555555ULL;
                t.v[3]=0x5555555555555555ULL;
        return t;
    }();
    return gt_u256(a, LIMIT) || eq_u256(a, LIMIT);
}

struct Brent256Res {
    bool cycle_found;
    uint64_t steps;
    uint64_t cycle_length;
    std::string cycle_value_dec; // for printing/log; derived at detection
    bool overflow;
};

static std::string u256_to_dec(u256 x){
    if (x.is_zero()) return "0";
    // long division by 10
    std::string out;
    while (!x.is_zero()){
        __uint128_t rem=0;
        for (int i=3;i>=0;i--){
            __uint128_t cur = (rem<<64) | x.v[i];
            x.v[i] = (uint64_t)(cur / 10);
            rem    = (uint64_t)(cur % 10);
        }
        out.push_back('0' + (int)rem);
    }
    std::reverse(out.begin(), out.end());
    return out;
}

static Brent256Res detect_cycle_brent_256_host(u256 n, uint64_t max_steps) {
    Brent256Res r{false,0,0,"",false};
    u256 tortoise = n, hare = n;
    uint64_t power=1, lambda=1, steps=0;

    while (steps < max_steps) {
        if (hare.is_one()) { r.steps=steps; return r; }

        if (hare.is_even()){
            int sh = ctz_u256(hare);
            hare = shr_u256(hare, sh);
            steps += sh;
        } else {
            if (would_overflow_3n_plus_1_u256(hare)){ r.overflow=true; r.steps=steps; return r; }
            u256 t = add_u256_u64(mul_u256_u64(hare,3), 1);
            int k = ctz_u256(t);
            hare = shr_u256(t, k);
            steps += 1 + k;
        }

        if (eq_u256(tortoise, hare)){
            r.cycle_found = true;
            r.cycle_length = lambda;
            r.steps = steps;
            r.cycle_value_dec = u256_to_dec(hare);
            return r;
        }

        if (lambda == power){
            tortoise = hare;
            power <<= 1;
            lambda = 0;
        }
        lambda++;
    }
    r.steps=steps; r.overflow=true;
    return r;
}

// ---------------- Host: align, counting, and printing ----------------
static void align_task_range(u128 base, uint64_t start_offset, uint64_t count,
                             u128& start, u128& end, uint64_t& delta)
{
    start = base + (u128)start_offset;
    end   = start + (u128)count;
    align_start_and_delta(start, delta);
}

// Cycle record for final printing
struct CycleRec {
    u128 seed;
    uint64_t len;
    // for 128-bit cycles: meet value in u128; for 256-bit we'll store decimal string
    bool is256;
    u128 meet128;
    std::string meet256_dec;
};

// ---------------- Main ----------------
int main(int argc, char** argv){
    MPI_Init(&argc, &argv);
    int rank=0, size=1;
    MPI_Comm_rank(MPI_COMM_WORLD,&rank);
    MPI_Comm_size(MPI_COMM_WORLD,&size);

    if (argc < 3){
        if (rank==0){
            fprintf(stderr,"Usage: %s <start_offset> <count> [--small-limit <bits>] [--tag <name>] [--no-print-cycles]\n", argv[0]);
        }
        MPI_Finalize(); return 1;
    }

    uint64_t start_offset = std::stoull(argv[1]);
    uint64_t count        = std::stoull(argv[2]);
    uint32_t SMALL_LIMIT_BITS = 20;
    const char* run_tag   = "v15cuda";
    bool print_cycles = true;

    for (int i=3;i<argc;i++){
        if (!strcmp(argv[i],"--small-limit") && i+1<argc) SMALL_LIMIT_BITS = std::stoul(argv[++i]);
        else if (!strcmp(argv[i],"--tag") && i+1<argc)     run_tag = argv[++i];
        else if (!strcmp(argv[i],"--no-print-cycles"))     print_cycles = false;
    }

    // pick device per-rank
    int dev_count=1;
    hipGetDeviceCount(&dev_count);
    hipSetDevice(rank % std::max(1,dev_count));

    // Build minimal memo and upload
    const uint64_t small_limit = (1ULL << SMALL_LIMIT_BITS);
    std::vector<uint32_t> memo(small_limit);
    build_min_memo(memo);

    uint32_t* d_memo=nullptr;
    hipMalloc(&d_memo, small_limit*sizeof(uint32_t));
    hipMemcpy(d_memo, memo.data(), small_limit*sizeof(uint32_t), hipMemcpyHostToDevice);
    DevMemo M{d_memo, small_limit};

    // MPI slice
    uint64_t count_per = count/size;
    uint64_t start_r   = start_offset + (uint64_t)rank * count_per;
    uint64_t count_r   = (rank==size-1) ? (count - (uint64_t)rank*count_per) : count_per;

    // Task range near 2^71
    const u128 base = (u128)1 << 71;
    u128 start_u128, end_u128;
    uint64_t delta=4;
    align_task_range(base, start_r, count_r, start_u128, end_u128, delta);

    if (rank==0){
        fprintf(stderr,"[MPI] size=%d\n", size);
    }

    // Launch params (tune as needed)
    const int BLOCK=256;
    const int GRID = 2048; // ~524,288 threads
    const size_t NTHREADS = (size_t)BLOCK * GRID;

    // Device accumulators
    ThreadAcc* d_acc=nullptr;
    hipMalloc(&d_acc, NTHREADS*sizeof(ThreadAcc));

    // Device flag queue
    const unsigned int QCAP = 2'000'000; // capacity for flagged seeds per rank
    FlaggedSeed* d_q=nullptr;
    hipMalloc(&d_q, QCAP*sizeof(FlaggedSeed));
    // reset queue counter
    hipMemset(&g_q_count, 0, sizeof(unsigned int));

    // Barrier & timing
    MPI_Barrier(MPI_COMM_WORLD);
    double t0 = MPI_Wtime();

    // Launch kernel
    collatz_kernel<<<GRID,BLOCK>>>(start_u128, end_u128, delta, M, d_q, QCAP, d_acc);
    hipDeviceSynchronize();

    // Pull back accumulators
    std::vector<ThreadAcc> acc(NTHREADS);
    hipMemcpy(acc.data(), d_acc, NTHREADS*sizeof(ThreadAcc), hipMemcpyDeviceToHost);

    // Pull back queue size and contents
    unsigned int h_q_count = 0;
    hipMemcpyFromSymbol(&h_q_count, HIP_SYMBOL(g_q_count), sizeof(unsigned int));
    if (h_q_count > QCAP) h_q_count = QCAP; // safety
    std::vector<FlaggedSeed> flagged(h_q_count);
    if (h_q_count) {
        hipMemcpy(flagged.data(), d_q, h_q_count*sizeof(FlaggedSeed), hipMemcpyDeviceToHost);
    }

    // Local aggregates
    unsigned long long tested_local=0, steps_local=0, over_local=0, fuse_local=0;
    for (const auto& a : acc){
        tested_local   += a.tested;
        steps_local    += a.steps_sum;
        over_local     += a.overflow_cnt;
        fuse_local     += a.fuse_cnt;
    }

    // Verification on host: Brent(128) and Brent(256) as needed
    std::vector<CycleRec> cycles_local;
    cycles_local.reserve(1024);

    for (const auto& fs : flagged){
        // First, Brent(128)
        Brent128Res r128 = detect_cycle_brent_128_host(fs.n, EXTENDED_FUSE);
        if (r128.cycle_found){
            CycleRec c;
            c.seed = fs.n;
            c.len  = r128.cycle_length;
            c.is256 = false;
            c.meet128 = r128.meet_value;
            cycles_local.push_back(c);
            continue;
        }
        if (r128.overflow){
            // Promote to 256
            u256 seed256(fs.n);
            Brent256Res r256 = detect_cycle_brent_256_host(seed256, EXTENDED_FUSE);
            if (r256.cycle_found){
                CycleRec c;
                c.seed = fs.n;
                c.len  = r256.cycle_length;
                c.is256 = true;
                c.meet256_dec = r256.cycle_value_dec;
                cycles_local.push_back(c);
            }
            // else: either overflowed 256 or timed out → no cycle confirmed
        }
        // else: verified OK within 128 without cycle (fuse false positive)
    }

    // Timing
    MPI_Barrier(MPI_COMM_WORLD);
    double elapsed = MPI_Wtime() - t0;

    // MPI reductions
    double elapsed_max=0.0;
    MPI_Reduce(&elapsed, &elapsed_max, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

    unsigned long long tested_sum=0, steps_sum=0, over_sum=0, fuse_sum=0;
    MPI_Reduce(&tested_local, &tested_sum, 1, MPI_UNSIGNED_LONG_LONG, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(&steps_local,  &steps_sum,  1, MPI_UNSIGNED_LONG_LONG, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(&over_local,   &over_sum,   1, MPI_UNSIGNED_LONG_LONG, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(&fuse_local,   &fuse_sum,   1, MPI_UNSIGNED_LONG_LONG, MPI_SUM, 0, MPI_COMM_WORLD);

    // Gather cycles to rank 0 for printing once
    // We'll serialize as plain text lines to keep it simple.
    // First, build local string
    std::string local_cycles_blob;
    {
        for (const auto& c : cycles_local){
            local_cycles_blob += (c.is256 ? "256 " : "128 ");
            local_cycles_blob += to_string_u128(c.seed);
            local_cycles_blob += " ";
            local_cycles_blob += std::to_string(c.len);
            local_cycles_blob += " ";
            if (c.is256) {
                local_cycles_blob += c.meet256_dec;
            } else {
                local_cycles_blob += to_string_u128(c.meet128);
            }
            local_cycles_blob += "\n";
        }
    }
    // MPI gather counts then data
    int local_len = (int)local_cycles_blob.size();
    std::vector<int> all_lens;
    if (rank==0) all_lens.resize(size);
    MPI_Gather(&local_len, 1, MPI_INT, rank==0?all_lens.data():nullptr, 1, MPI_INT, 0, MPI_COMM_WORLD);

    std::string all_blob;
    if (rank==0){
        size_t total=0;
        for (int i=0;i<size;i++) total += all_lens[i];
        all_blob.resize(total);
    }

    // displacements
    std::vector<int> displs;
    if (rank==0){
        displs.resize(size);
        int off=0;
        for (int i=0;i<size;i++){ displs[i]=off; off += all_lens[i]; }
    }

    MPI_Gatherv(local_cycles_blob.data(), local_len, MPI_CHAR,
                rank==0?all_blob.data():nullptr,
                rank==0?all_lens.data():nullptr,
                rank==0?displs.data():nullptr,
                MPI_CHAR, 0, MPI_COMM_WORLD);

    // Rank 0 prints once
    if (rank==0){
        double thr = tested_sum / std::max(1e-9, elapsed_max);
        double avg_steps = tested_sum ? (double)steps_sum / (double)tested_sum : 0.0;

        std::cout << "\n=== V1.5-cuda (MPI+CUDA) Summary ===\n";
        std::cout << "Tag:               " << run_tag << "\n";
        std::cout << "MPI ranks:         " << size << "\n";
        std::cout << "Tested:            " << tested_sum << " numbers\n";
        std::cout << "Time (max rank):   " << (uint64_t)(elapsed_max*1000) << " ms\n";
        std::cout << "Throughput:        " << (uint64_t)thr << " nums/sec\n";
        std::cout << "Avg steps:         " << avg_steps << "\n";
        std::cout << "Overflows (fast):  " << over_sum << "\n";
        std::cout << "Fuse hits (fast):  " << fuse_sum << "\n";

        // Cycle print at the end (single place)
        if (!all_blob.empty()){
            // Count lines
            size_t cnt = std::count(all_blob.begin(), all_blob.end(), '\n');
            std::cout << "Cycles found:      " << cnt << "\n";
            if (print_cycles){
                std::cout << "--- CYCLES (format: <bits> <seed> <length> <meet_value>) ---\n";
                std::cout << all_blob;
            }
        } else {
            std::cout << "Cycles found:      0\n";
        }
        std::cout << "=== SUCCESS ===\n";
    }

    // Cleanup
    hipFree(d_q);
    hipFree(d_acc);
    hipFree(d_memo);

    MPI_Finalize();
    return 0;
}
